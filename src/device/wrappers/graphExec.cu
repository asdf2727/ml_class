#include "graphExec.cuh"

void device::graphExec::update (const device::graph &new_graph)  {
	if (data != nullptr) {
		hipGraphExecUpdateResult updateResult;
		hipGraphNode_t errorNode;
		cudaTry(hipGraphExecUpdate(data, new_graph, &errorNode, &updateResult));
		if (updateResult != hipGraphExecUpdateSuccess) {
			cudaTry(hipGraphExecDestroy(data));
			data = nullptr;
		}
	}
	if (data == nullptr) {
		cudaTry(hipGraphInstantiate(&data, new_graph, nullptr, nullptr, 0));
	}
}