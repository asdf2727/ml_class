#include "linearLayerBatched.cuh"

#include <hiprand.h>

void device::linearLayerBatched::buildForward (device::graph *&fwd) {
	hipGraphCreate((hipGraph_t*)fwd, 0);
	hipStream_t stream;
	hipStreamCreate(&stream);
	hipStreamBeginCapture(stream, hipStreamCaptureModeThreadLocal);
	hipblasHandle_t virtual_handle;
	hipblasCreate(&virtual_handle);

	hipblasSetStream(virtual_handle, stream);
	// output[out*batch] = trans(mult)[out*(in+1)] * input[(in+1)*batch]
	hipblasSgemm(virtual_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, out_size, batch_size, in_size + 1,
	            &const1, mul, mul.pitch, input->val, input->val.pitch,
	            &const0, output->val, output->val.pitch);

	hipblasDestroy(virtual_handle);
	hipStreamEndCapture(stream, (hipGraph_t*)fwd);
	hipStreamDestroy(stream);
}

void device::linearLayerBatched::buildBackward (device::graph *&back) {
	hipGraphCreate((hipGraph_t*)back, 0);
	hipStream_t stream1, stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamBeginCapture(stream1, hipStreamCaptureModeThreadLocal);
	hipblasHandle_t virtual_handle;
	hipblasCreate(&virtual_handle);

	hipEvent_t event1 = nullptr;
	hipEventCreate(&event1);
	hipEventRecord(event1, stream1);
	hipStreamWaitEvent(stream2, event1);

	hipblasSetStream(virtual_handle, stream1);
	// mult_der[(in+1)*out] += input[(in+1)*batch] * trans(output_der)[batch*out]
	hipblasSgemm(virtual_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, in_size + 1, out_size, batch_size,
	            &const1, input->val, input->val.pitch, *output->der, output->der->pitch,
	            &const1, *mul_der, mul_der->pitch);

	hipblasSetStream(virtual_handle, stream2);
	// input_der[in*batch] = mult[in*out] * output_der[out*batch]
	hipblasSgemm(virtual_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, in_size, batch_size, out_size, &const1,
	            mul, mul.pitch, *output->der, output->der->pitch,
	            &const0, *input->der, input->der->pitch);

	hipEvent_t event2 = nullptr;
	hipEventCreate(&event2);
	hipEventRecord(event2, stream2);
	hipStreamWaitEvent(stream1, event2);

	hipblasDestroy(virtual_handle);
	hipStreamEndCapture(stream1, (hipGraph_t*)back);
	hipEventDestroy(event1);
	hipEventDestroy(event2);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
}

void device::linearLayerBatched::buildDescent (device::graph *&desc) {
	hipStream_t stream;
	hipStreamCreate(&stream);
	hipStreamBeginCapture(stream, hipStreamCaptureModeThreadLocal);
	hipblasHandle_t virtual_handle;
	hipblasCreate(&virtual_handle);

	hipblasSetStream(virtual_handle, stream);
	// mult[(in+1)*out] = mult[(in+1)*out] + step_size * mult_der_sum[(in+1)*out]
	hipblasSgeam(virtual_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, in_size + 1, out_size,
	            &const1, mul, mul.pitch,
	            &step_size, *mul_der, mul_der->pitch, mul, mul.pitch);
	mul_der->set(0x00, stream);

	hipblasDestroy(virtual_handle);
	hipStreamEndCapture(stream, (hipGraph_t*)desc);
	hipStreamDestroy(stream);
}

void device::linearLayerBatched::resetWeights (const float mean, const float std_dev, const unsigned long long seed)  {
	hiprandGenerator_t eng;
	hiprandCreateGenerator(&eng, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(eng, seed);
	hiprandGenerateNormal(eng, mul, batch_size * mul.pitch, mean, std_dev);
	hiprandDestroyGenerator(eng);
	mul_der->set(0x00);
}

void device::linearLayerBatched::loadWeights (const std::vector <float> &weights) {
	assert(weights.size() == (in_size + 1) * out_size);
	mul = weights.data();
	mul_der->set(0x00);
}

inline std::vector <float> device::linearLayerBatched::saveWeights () const {
	std::vector <float> ans((in_size + 1) * out_size);
	mul.toHost(ans.data());
	return ans;
}
