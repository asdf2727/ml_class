#include "hip/hip_runtime.h"
#include "activateLayerBatched.cuh"

__global__ void devActivateMatrix (float *mat, const size_t pitch, const size_t width,
                                   const size_t height, float (*fwdFunc) (float val)) {
	const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const size_t idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx < width && idy < height) { mat[idx + pitch * idy] = fwdFunc(mat[idx + pitch * idy]); }
}

void device::activateLayerBatched::buildForward (device::graph *&fwd) {
	hipStream_t stream;
	hipStreamCreate(&stream);
	hipStreamBeginCapture(stream, hipStreamCaptureModeThreadLocal);

	dim3 blockDim(16, 16);
	dim3 gridDim(calcBlocks(data->getSize(), blockDim.x), calcBlocks(getBatchSize(), blockDim.y));
	devActivateMatrix<<<gridDim, blockDim, 0, stream>>>(data->val, data->val.pitch, data->getSize(),
	                                                    getBatchSize(), act.fwdFunc);

	hipStreamEndCapture(stream, (hipGraph_t*)fwd);
	hipStreamDestroy(stream);
}

__global__ void devActivateMatrixDer (const size_t width, const size_t height,
                                      const float *val, const size_t val_pitch,
                                      float *der, const size_t der_pitch,
                                      float (*backFunc) (float act)) {
	const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const size_t idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx < width && idy < height) {
		der[idx + der_pitch * idy] *= backFunc(val[idx + val_pitch * idy]);
	}
}

void device::activateLayerBatched::buildBackward (device::graph *&back) {
	hipStream_t stream;
	hipStreamCreate(&stream);
	hipStreamBeginCapture(stream, hipStreamCaptureModeThreadLocal);

	dim3 blockDim(16, 16);
	dim3 gridDim(calcBlocks(data->getSize(), blockDim.x), calcBlocks(getBatchSize(), blockDim.y));
	devActivateMatrixDer<<<gridDim, blockDim, 0, stream>>>(data->getSize(), getBatchSize(),
	                                                    data->val, data->val.pitch,
	                                                    *data->der, data->der->pitch,
	                                                       act.backFunc);

	hipStreamEndCapture(stream, (hipGraph_t*)back);
	hipStreamDestroy(stream);
}

void device::activateLayerBatched::changeData (device::neuronArrayBatched &new_data) {
	assert(data->getSize() == new_data.getBatchSize() && data->getSize() == new_data.getBatchSize());
	data = new_data;
	fwd.invalidate();
	back.invalidate();
}
